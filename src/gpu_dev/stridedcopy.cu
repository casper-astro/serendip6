#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>

int main(int argc, char* argv[])
{
    char* p = NULL;
    char* q = NULL;
    char* r = NULL;
    int i = 0;
    hipError_t iRet;

    p = (char*) malloc(100);
    assert(p != NULL);
    q = (char*) malloc(20);
    assert(q != NULL);
    r = (char*) malloc(40);
    assert(r != NULL);

    for (i = 0; i < 100; ++i)
    {
        p[i] = i;
    }

    iRet = hipMemcpy2D(q, 1, p, 5, 1, 20, hipMemcpyHostToHost);
    printf("**********\niRet = %d\n**********\n", iRet);
    iRet = hipMemcpy2D(r, 2, p, 5, 2, 20, hipMemcpyHostToHost);
    printf("**********\niRet = %d\n**********\n", iRet);

    for (i = 0; i < 100; ++i)
    {
        printf("%d ", p[i]);
    }
    printf("\n**********\n");
    for (i = 0; i < 20; ++i)
    {
        printf("%d ", q[i]);
    }
    printf("\n**********\n");
    for (i = 0; i < 40; ++i)
    {
        printf("%d ", r[i]);
    }

    printf("\n");

    free(r);
    free(q);
    free(p);

    return 0;
}

