#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>

int main(int argc, char* argv[])
{
    int* p = NULL;
    int* q = NULL;
    int i = 0;
    hipError_t iRet;

    p = (int*) malloc(8*64*sizeof(int));
    assert(p != NULL);
    q = (int*) malloc(2*64*sizeof(int));
    assert(q != NULL);

    for (i = 0; i < 8*64; ++i)
    {
        p[i] = i;
    }

    iRet = hipMemcpy2D(q, 2 * sizeof(int), p, 8 * sizeof(int), 2 * sizeof(int), 64, hipMemcpyHostToHost);
    printf("**********\niRet = %d\n**********\n", iRet);

    for (i = 0; i < 8*64; ++i)
    {
        printf("%d ", p[i]);
    }
    printf("\n**********\n");
    for (i = 0; i < 2*64; ++i)
    {
        printf("%d ", q[i]);
    }

    printf("\n");

    free(q);
    free(p);

    return 0;
}

